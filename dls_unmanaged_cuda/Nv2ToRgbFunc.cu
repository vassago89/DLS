#include "hip/hip_runtime.h"
#pragma once

#include "Nv2ToRgbFunc.h"

#ifndef max
#define max(a,b)  (((a) > (b)) ? (a) : (b))
#endif



#ifndef min
#define min(a,b)  (((a) < (b)) ? (a) : (b))
#endif

namespace dlsunmanaged {
	namespace cuda {
		__constant__ float constHueColorSpaceMat[9] = 
		{ 
			1.164f, 0.0f, 1.596f, 
			1.164f, -0.391f, -0.813f, 
			1.164f, 2.018f, 0.0f 
		};

		__global__ void NV12ToRGB(
			unsigned char* srcY, 
			unsigned char* srcUV,
			size_t nSourcePitch,
			unsigned char* dstImage, 
			size_t nDestPitch,
			unsigned int width,
			unsigned int height)
		{
			const int x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
			const int y = blockIdx.y * (blockDim.y << 1) + (threadIdx.y << 1);

			if (x >= width || y >= height)
				return;

			const int y_chroma = y >> 1;

			float y1 = srcY[y * nSourcePitch + x] - 16;
			float y2 = srcY[y * nSourcePitch + x + 1] - 16;
			float y3 = srcY[(y + 1) * nSourcePitch + x] - 16;
			float y4 = srcY[(y + 1) * nSourcePitch + x + 1] - 16;

			float cb  = srcUV[y_chroma * nSourcePitch + x] - 128;
			float cr = srcUV[y_chroma * nSourcePitch + x + 1] - 128;

			int index = y * nDestPitch + x * 3;
			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[6] * y1
					+ constHueColorSpaceMat[7] * cb
					+ constHueColorSpaceMat[8] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[3] * y1
					+ constHueColorSpaceMat[4] * cb
					+ constHueColorSpaceMat[5] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[0] * y1
					+ constHueColorSpaceMat[1] * cb
					+ constHueColorSpaceMat[2] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[6] * y2
					+ constHueColorSpaceMat[7] * cb
					+ constHueColorSpaceMat[8] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[3] * y2
					+ constHueColorSpaceMat[4] * cb
					+ constHueColorSpaceMat[5] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[0] * y2
					+ constHueColorSpaceMat[1] * cb
					+ constHueColorSpaceMat[2] * cr));

			index = (y + 1) * nDestPitch + x * 3;
			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[6] * y3
					+ constHueColorSpaceMat[7] * cb
					+ constHueColorSpaceMat[8] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[3] * y3
					+ constHueColorSpaceMat[4] * cb
					+ constHueColorSpaceMat[5] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[0] * y3
					+ constHueColorSpaceMat[1] * cb
					+ constHueColorSpaceMat[2] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[6] * y4
					+ constHueColorSpaceMat[7] * cb
					+ constHueColorSpaceMat[8] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[3] * y4
					+ constHueColorSpaceMat[4] * cb
					+ constHueColorSpaceMat[5] * cr));

			dstImage[index++] =
				max(0, min(255, constHueColorSpaceMat[0] * y4
					+ constHueColorSpaceMat[1] * cb
					+ constHueColorSpaceMat[2] * cr));
		}

		void Nv2ToRgbFunc(
			unsigned char* srcY,
			unsigned char* srcUV,
			int srcPitch, 
			unsigned char* dest,
			int destPitch, 
			int width,
			int height)
		{
			// Final Stage: NV12toARGB color space conversion
			//checkCudaError(hipDeviceSynchronize());

			/*unsigned char* src1 = cuda::CudaAlloc<unsigned char>(srcPitch, height);
			unsigned char* src2 = cuda::CudaAlloc<unsigned char>(srcPitch, height / 2);

			cuda::CudaCopyDevToDev(srcY, src1, srcPitch * height);
			cuda::CudaCopyDevToDev(srcUV, src2, srcPitch * height / 2);*/

			dim3 block(32, 8);
			dim3 grid((width + (2 * block.x) - 1) / (2 * block.x), (height + (2 * block.y) - 1) / (2 * block.y));

			NV12ToRGB << <grid, block, 0 >> > (
				srcY,
				srcUV,
				srcPitch,
				dest,
				destPitch,
				width,
				height);

			checkCudaError(hipGetLastError());
			checkCudaError(hipDeviceSynchronize());
		}

		void CudaInit()
		{
			hipDeviceReset();

			int count = 0;
			checkCudaError(hipGetDeviceCount(&count));

			int gpu = 0;
			checkCudaError(hipSetDevice(gpu));
		}

		void CudaRelease()
		{
			hipDeviceReset();
		}

		
	}
}